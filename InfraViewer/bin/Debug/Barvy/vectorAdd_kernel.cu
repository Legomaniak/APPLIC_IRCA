
#include <hip/hip_runtime.h>
﻿/**
* CUDA Kernel Device code
*
* Computes the vector addition of A and B into C. The 3 vectors have the same
* number of elements numElements.
*/

extern "C" __global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		C[i] = A[i] + B[i];
	}
}

/**
* CUDA Kernel Device code
*
* Decode color value from array A by color patern array B to byte array C.
*/
extern "C" __global__ void vectorObarvi(const int *A, const unsigned char *B, unsigned char *C, int numElements)//480*640,655535,640*480*4,640*480
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < numElements)
	{
		int D = A[i] * 4;
		i *= 4;
		for (int j = 0; j < 4; j++)
		{
			C[i + j] = B[D + j];
		}
	}
}
/**
* CUDA Kernel Device code
*
* Decode color value from array A by color patern array B to byte array C. Decoded value of A in D.
*/
extern "C" __global__ void vectorObarviSource(const unsigned char *A, const unsigned char *B, unsigned char *C, int *D, int numElements)//480*640*4,655535,640*480*4,640*480
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	int ic = i * 4;

	if (i < numElements)
	{
		D[i] = (int)(A[ic + 1] << 8 | A[ic]);
		int k = D[i] * 4;
		//int D = (A[i + 3] << 24 | A[i + 2] << 16 | A[i + 1] << 8 | A[i]) * 4;
		for (int j = 0; j < 4; j++)
		{
			C[ic + j] = B[k + j];
		}
	}
}
